#include "hip/hip_runtime.h"
#include <vector>
#include <chrono>
#include "smith_waterman.hpp"
#include "constants.hpp"

namespace {

constexpr std::size_t THREADS_PER_BLOCK = 256;

struct Score {
    int score{};
    int row{};
    int col{};
};

__device__ Score dMaxScore;

__global__ void computeWavefrontDiag(
    int* __restrict__ currH,
    const int* __restrict__ prevH,
    const int* __restrict__ prevPrevH,
    int* __restrict__ currE,
    int* __restrict__ currF,
    const int* __restrict__ prevF,
    Direction* __restrict__ trace,
    const char* __restrict__ seq1,
    const char* __restrict__ seq2,
    std::size_t seq1Size,
    std::size_t seq2Size,
    std::size_t k
) {
    __shared__ Score warpMax[THREADS_PER_BLOCK / 32];

    std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    std::size_t startRow = (k < seq2Size) ? 0 : k - seq2Size + 1;
    std::size_t row = startRow + i;
    std::size_t col = k - row;
    if (row >= seq1Size || col >= seq2Size) return;

    int offset = row + col - seq2Size;
    offset = (offset < -1) ? -1 : (offset > 1 ? 1 : offset);

    int diagI = i + offset;
    int leftI = i + (offset >= 0);
    int upI = leftI - 1;

    int diag = (diagI >= 0 ? prevPrevH[diagI] : 0) + (seq1[row] == seq2[col] ? MATCH : MISMATCH);
    currE[i] = max(prevH[leftI] + GAP_OPEN, currE[leftI] + GAP_EXTEND);
    currF[i] = upI >= 0 ? max(prevH[upI] + GAP_OPEN, prevF[upI] + GAP_EXTEND) : max(GAP_OPEN, GAP_EXTEND);
    currH[i] = max(max(0, diag), max(currE[i], currF[i]));

    std::size_t traceIdx = row * seq2Size + col;
    if (currH[i] == diag) [[likely]] {
        trace[traceIdx] = Direction::Diag;
    } else if (currH[i] == currE[i]) {
        trace[traceIdx] = Direction::Left;
    } else if (currH[i] == currF[i]) {
        trace[traceIdx] = Direction::Up;
    } else [[unlikely]] {
        trace[traceIdx] = Direction::None;
    }

    Score localMax = {currH[i], static_cast<int>(row), static_cast<int>(col)};
    int laneId = threadIdx.x % warpSize;
    unsigned mask = 0xffffffff;
    
    for (int offset = 16; offset > 0; offset /= 2) {
        int otherScore = __shfl_xor_sync(mask, localMax.score, offset);
        int otherRow = __shfl_xor_sync(mask, localMax.row, offset);
        int otherCol = __shfl_xor_sync(mask, localMax.col, offset);
        if (otherScore > localMax.score) {
            localMax.score = otherScore;
            localMax.row = otherRow;
            localMax.col = otherCol;
        }
    }
    
    int warpId = threadIdx.x / 32;
    
    if (laneId == 0) {
        warpMax[warpId] = localMax;
    }
    __syncthreads();
    
    if (threadIdx.x < THREADS_PER_BLOCK / 32) {
        localMax = warpMax[threadIdx.x];
    }
    __syncthreads();
    
    if (threadIdx.x < 32) {
        for (int offset = 16; offset > 0; offset /= 2) {
            int otherScore = __shfl_xor_sync(mask, localMax.score, offset);
            int otherRow = __shfl_xor_sync(mask, localMax.row, offset);
            int otherCol = __shfl_xor_sync(mask, localMax.col, offset);
            if (otherScore > localMax.score) {
                localMax.score = otherScore;
                localMax.row = otherRow;
                localMax.col = otherCol;
            }
        }
    }
    
    if (threadIdx.x == 0) {
        int old = atomicMax(&dMaxScore.score, localMax.score);
        if (localMax.score > old) {
            atomicExch(&dMaxScore.row, localMax.row);
            atomicExch(&dMaxScore.col, localMax.col);
        }
    }
}

}

SmithWaterman::AlignmentReport SmithWaterman::cuda(const Fasta& fasta1, const Fasta& fasta2) {
    hipFree(0);
    auto startTime = std::chrono::high_resolution_clock::now();

    const std::size_t maxDiagLen = min(fasta1.size, fasta2.size);
    const std::size_t diagNum = fasta1.size + fasta2.size - 1;

    char *dSeq1, *dSeq2;

    hipMalloc(&dSeq1, fasta1.size * sizeof(char));
    hipMalloc(&dSeq2, fasta2.size * sizeof(char));

    hipMemcpy(dSeq1, fasta1.sequence.data(), fasta1.size * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dSeq2, fasta2.sequence.data(), fasta2.size * sizeof(char), hipMemcpyHostToDevice);

    int *dCurrH, *dPrevH, *dPrevPrevH, *dCurrE, *dCurrF, *dPrevF;
    Direction* dTrace;

    hipMalloc(&dCurrH, maxDiagLen * sizeof(int));
    hipMalloc(&dPrevH, maxDiagLen * sizeof(int));
    hipMalloc(&dPrevPrevH, maxDiagLen * sizeof(int));
    hipMalloc(&dCurrE, maxDiagLen * sizeof(int));
    hipMalloc(&dCurrF, maxDiagLen * sizeof(int));
    hipMalloc(&dPrevF, maxDiagLen * sizeof(int));
    hipMalloc(&dTrace, fasta1.size * fasta2.size * sizeof(Direction));

    hipMemset(dCurrH, 0, maxDiagLen * sizeof(int));
    hipMemset(dPrevH, 0, maxDiagLen * sizeof(int));
    hipMemset(dPrevPrevH, 0, maxDiagLen * sizeof(int));
    hipMemset(dCurrE, 0, maxDiagLen * sizeof(int));
    hipMemset(dCurrF, 0, maxDiagLen * sizeof(int));
    hipMemset(dPrevF, 0, maxDiagLen * sizeof(int));

    dim3 gridDim((maxDiagLen + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    dim3 blockDim(THREADS_PER_BLOCK);
    int* temp;

    for (std::size_t k = 0; k < diagNum; ++k) {
        computeWavefrontDiag<<<gridDim, blockDim>>>(
            dCurrH, dPrevH, dPrevPrevH, dCurrE, dCurrF, dPrevF,
            dTrace, dSeq1, dSeq2, fasta1.size, fasta2.size, k
        );
        hipDeviceSynchronize();
    
        temp = dPrevPrevH;
        dPrevPrevH = dPrevH;
        dPrevH = dCurrH;
        dCurrH = temp;
        temp = dPrevF;
        dPrevF = dCurrF;
        dCurrF = temp;
    }

    std::vector<Direction> hTrace(fasta1.size * fasta2.size);
    hipMemcpy(hTrace.data(), dTrace, hTrace.size() * sizeof(Direction), hipMemcpyDeviceToHost);

    Score hMaxScore;
    hipMemcpyFromSymbol(&hMaxScore, HIP_SYMBOL(dMaxScore), sizeof(Score), 0, hipMemcpyDeviceToHost);

    int maxScore = hMaxScore.score;
    std::pair<std::size_t, std::size_t> maxPos{static_cast<std::size_t>(hMaxScore.row + 1), static_cast<std::size_t>(hMaxScore.col + 1)};

    hipFree(dSeq1);
    hipFree(dSeq2);
    hipFree(dCurrH);
    hipFree(dPrevH);
    hipFree(dPrevPrevH);
    hipFree(dCurrE);
    hipFree(dCurrF);
    hipFree(dPrevF);
    hipFree(dTrace);

    auto endTime = std::chrono::high_resolution_clock::now();
    auto result = traceback(fasta1, fasta2, maxPos, [&](std::size_t i, std::size_t j) {
        return hTrace[(i - 1) * fasta2.size + (j - 1)];
    });
    return {result, std::chrono::duration<double, std::milli>(endTime - startTime).count(), maxScore};
}
